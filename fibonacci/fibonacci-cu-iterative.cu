#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void fib_iter(long *rezultat, int n) {
    long a = 0, b = 1;
    for (int i = 2; i <= n; i++) {
        long temp = a;
        a = b;
        b = temp + b;
    }
    *rezultat = a;
}

int main() {
    int n = 40;
    long h_rezultat;
    long *d_rezultat;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&d_rezultat, sizeof(long));

    hipEventRecord(start);
    fib_iter<<<1024, 16>>>(d_rezultat, n);
    hipEventRecord(stop);

    hipMemcpy(&h_rezultat, d_rezultat, sizeof(long), hipMemcpyDeviceToHost);
    hipFree(d_rezultat);

    hipEventSynchronize(stop);
    float vreme = 0;
    hipEventElapsedTime(&vreme, start, stop);

    printf("Fibonacci od %d je %ld\n", n, h_rezultat);
    printf("Vreme izvršenja: %.2f ms\n", vreme);
    return 0;
}
