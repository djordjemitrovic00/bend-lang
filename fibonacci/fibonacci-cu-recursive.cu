#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int fib(int n) {
    if (n <= 1)
        return n;
    else
        return fib(n - 1) + fib(n - 2);
}

__global__ void fibonacci(int *rezultat, int n) {
    *rezultat = fib(n);
}

int main() {
    int n = 40;
    int h_rezultat;
    int *d_rezultat;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&d_rezultat, sizeof(int));

    hipEventRecord(start);
    fibonacci<<<1024, 16>>>(d_rezultat, n);
    hipEventRecord(stop);

    hipMemcpy(&h_rezultat, d_rezultat, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_rezultat);

    hipEventSynchronize(stop);
    float vreme = 0;
    hipEventElapsedTime(&vreme, start, stop);

    printf("Fibonacci od %d je %d\n", n, h_rezultat);
    printf("Vreme izvršenja: %.2f ms\n", vreme);
    return 0;
}
