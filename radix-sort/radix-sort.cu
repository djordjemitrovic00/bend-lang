#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void counting_sort(int *arr, int *output, int *count, int n, int exp) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        atomicAdd(&count[(arr[i] / exp) % 10], 1);
    }
    __syncthreads();
    for (int i = 1; i < 10; i++) {
        count[i] += count[i - 1];
    }
    __syncthreads();
    if (i < n) {
        int pos = atomicSub(&count[(arr[i] / exp) % 10], 1);
        output[pos - 1] = arr[i];
    }
}

void radix_sort(int *arr, int n) {
    int *d_arr, *d_output, *d_count;
    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_output, n * sizeof(int));
    hipMalloc((void **)&d_count, 10 * sizeof(int));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int max = arr[0];
    for (int i = 1; i < n; i++) {
        if (arr[i] > max) max = arr[i];
    }

    int exp = 1;
    while (max / exp > 0) {
        hipMemset(d_count, 0, 10 * sizeof(int));

        counting_sort<<<(n + 255) / 256, 256>>>(d_arr, d_output, d_count, n, exp);
        hipMemcpy(d_arr, d_output, n * sizeof(int), hipMemcpyDeviceToDevice);
        exp *= 10;
    }

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_arr);
    hipFree(d_output);
    hipFree(d_count);
}

int main() {
    int n = 20;
    int *arr = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        arr[i] = rand() % n;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    radix_sort(arr, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Vreme izvršenja: %f ms\n", milliseconds);

    free(arr);
    return 0;
}
