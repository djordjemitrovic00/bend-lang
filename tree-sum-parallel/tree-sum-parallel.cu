#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

struct MyTree {
    int val;
    MyTree *left, *right;
};

__device__ int sum_device(MyTree *tree) {
    if (tree == NULL) {
        return 0;
    } else {
        int left_sum = sum_device(tree->left);
        int right_sum = sum_device(tree->right);
        return tree->val + left_sum + right_sum;
    }
}

__global__ void sum_kernel(MyTree *tree, int *result) {
    *result = sum_device(tree);
}

__global__ MyTree* gen_device(int depth, int val) {
    if (depth == 0) {
        return NULL;
    } else {
        MyTree *node = (MyTree *)malloc(sizeof(MyTree));
        node->val = val;
        node->left = gen_device(depth - 1, 2 * val);
        node->right = gen_device(depth - 1, 2 * val + 1);
        return node;
    }
}

int main() {
    int depth = 14;
    int *d_result, h_result;
    hipMalloc(&d_result, sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    MyTree *d_tree = NULL;
    gen_device<<<32, 512>>>(depth, 1);
    hipDeviceSynchronize();

    sum_kernel<<<32, 512>>>(d_tree, d_result);
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Rezultat: %d\n", h_result);
    printf("Vreme izvršenja: %f ms\n", milliseconds);

    hipFree(d_result);
    return 0;
}
