#include <stdio.h>
#include <hip/hip_runtime.h>

struct MyTree
{
    int val;
    int left_index;
    int right_index;
};

__global__ void gen_tree(MyTree *nodes, int depth)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_nodes = (1 << (depth + 1)) - 1;

    if (idx < num_nodes)
    {
        nodes[idx].val = idx + 1;

        int left_child = 2 * idx + 1;
        int right_child = 2 * idx + 2;

        if (left_child < num_nodes)
        {
            nodes[idx].left_index = left_child;
        }
        else
        {
            nodes[idx].left_index = -1;
        }

        if (right_child < num_nodes)
        {
            nodes[idx].right_index = right_child;
        }
        else
        {
            nodes[idx].right_index = -1;
        }
    }
}

__global__ void sum_tree(MyTree *nodes, int *result, int depth)
{
    __shared__ int shared_sum[1024];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_nodes = (1 << (depth + 1)) - 1;

    if (idx < num_nodes)
    {
        shared_sum[threadIdx.x] = nodes[idx].val;
    }
    else
    {
        shared_sum[threadIdx.x] = 0;
    }

    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        atomicAdd(result, shared_sum[0]);
    }
}

int main()
{
    int depth = 14;
    int num_nodes = (1 << (depth + 1)) - 1;
    int *d_result, h_result = 0;
    MyTree *d_tree;

    hipMalloc(&d_tree, num_nodes * sizeof(MyTree));
    hipMalloc(&d_result, sizeof(int));
    hipMemset(d_result, 0, sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gen_tree<<<(num_nodes + 1023) / 1024, 1024>>>(d_tree, depth);
    hipDeviceSynchronize();

    sum_tree<<<(num_nodes + 1023) / 1024, 1024>>>(d_tree, d_result, depth);
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Rezultat: %d\n", h_result);
    printf("Vreme izvršenja: %f ms\n", milliseconds);

    hipFree(d_tree);
    hipFree(d_result);
    return 0;
}
