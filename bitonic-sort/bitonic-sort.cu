#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define N 18
#define THREADS_PER_BLOCK 256

__device__ void swap(int *a, int *b)
{
    int temp = *a;
    *a = *b;
    *b = temp;
}

__device__ void bitonicMerge(int *data, int low, int cnt, int dir)
{
    if (cnt > 1)
    {
        int k = cnt / 2;
        for (int i = low; i < low + k; i++)
        {
            if (dir == (data[i] > data[i + k]))
            {
                swap(&data[i], &data[i + k]);
            }
        }
        bitonicMerge(data, low, k, dir);
        bitonicMerge(data, low + k, k, dir);
    }
}

__global__ void bitonicSortKernel(int *data, int j, int k)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    if (ixj > i)
    {
        if ((i & k) == 0)
        {
            if (data[i] > data[ixj])
            {
                swap(&data[i], &data[ixj]);
            }
        }
        else
        {
            if (data[i] < data[ixj])
            {
                swap(&data[i], &data[ixj]);
            }
        }
    }
}

void bitonicSort(int *data, int n)
{
    int *d_data;
    size_t size = n * sizeof(int);

    hipMalloc(&d_data, size);
    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

    dim3 blocks(N / THREADS_PER_BLOCK, 1);
    dim3 threads(THREADS_PER_BLOCK, 1);

    for (int k = 2; k <= N; k <<= 1)
    {
        for (int j = k >> 1; j > 0; j >>= 1)
        {
            bitonicSortKernel<<<blocks, threads>>>(d_data, j, k);
            hipDeviceSynchronize();
        }
    }

    hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
    hipFree(d_data);
}

int main()
{
    int *data = new int[N];

    for (int i = 0; i < N; i++)
    {
        data[i] = rand() % 1000;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    bitonicSort(data, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Vreme izvrsenja: " << elapsedTime << " ms" << std::endl;

    std::cout << "Sortiran niz: ";
    for (int i = 0; i < N; i++)
    {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;

    delete[] data;
    return 0;
}
