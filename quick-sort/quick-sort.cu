#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int partition(int *arr, int left, int right)
{
    int pivot = arr[right];
    int i = left - 1;
    for (int j = left; j < right; j++)
    {
        if (arr[j] <= pivot)
        {
            i++;
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }
    int temp = arr[i + 1];
    arr[i + 1] = arr[right];
    arr[right] = temp;
    return i + 1;
}

__global__ void quicksort_kernel(int *arr, int *lefts, int *rights, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        int left = lefts[idx];
        int right = rights[idx];
        if (left < right)
        {
            int pivot = partition(arr, left, right);

            if (pivot - 1 > left)
            {
                lefts[idx] = left;
                rights[idx] = pivot - 1;
            }
            else
            {
                lefts[idx] = pivot + 1;
                rights[idx] = right;
            }
        }
    }
}

void quicksort_host(int *arr, int n)
{
    int *d_arr, *d_lefts, *d_rights;
    int *lefts = (int *)malloc(n * sizeof(int));
    int *rights = (int *)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++)
    {
        lefts[i] = 0;
        rights[i] = n - 1;
    }

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_lefts, n * sizeof(int));
    hipMalloc((void **)&d_rights, n * sizeof(int));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lefts, lefts, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rights, rights, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int step = 0; step < n; step++)
    {
        quicksort_kernel<<<gridSize, blockSize>>>(d_arr, d_lefts, d_rights, n);
        hipDeviceSynchronize();
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Vreme izvršenja: %f ms\n", milliseconds);

    hipFree(d_arr);
    hipFree(d_lefts);
    hipFree(d_rights);
    free(lefts);
    free(rights);
}

int main()
{
    int n = 14;
    int *arr = (int *)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++)
    {
        arr[i] = rand() % n;
    }

    quicksort_host(arr, n);

    printf("Sortirani niz: ");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");

    free(arr);
    return 0;
}
