#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void quicksort(int *arr, int left, int right);

__device__ int partition(int *arr, int left, int right) {
    int pivot = arr[right];
    int i = left - 1;
    for (int j = left; j < right; j++) {
        if (arr[j] <= pivot) {
            i++;
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }
    int temp = arr[i + 1];
    arr[i + 1] = arr[right];
    arr[right] = temp;
    return i + 1;
}

__global__ void quicksort(int *arr, int left, int right) {
    if (left < right) {
        int pivot = partition(arr, left, right);

        if (pivot - left > 1024) {
            quicksort<<<32, 32>>>(arr, left, pivot - 1);
        } else {
            quicksort<<<32, 32>>>(arr, pivot + 1, right);
        }
    }
}

int main() {
    int n = 14;
    int *arr, *d_arr;
    arr = (int *)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        arr[i] = rand() % n;
    }

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    quicksort<<<32, 512>>>(d_arr, 0, n - 1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Vreme izvršenja: %f ms\n", milliseconds);

    hipFree(d_arr);
    free(arr);
    return 0;
}
