#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void bubble_sort(int *arr, int n) {
    for (int i = 0; i < n; i++) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n - i - 1 && arr[idx] > arr[idx + 1]) {
            int temp = arr[idx];
            arr[idx] = arr[idx + 1];
            arr[idx + 1] = temp;
        }
        __syncthreads();
    }
}

int main() {
    int n = 100;
    int *h_arr = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        h_arr[i] = rand() % 100;
    }

    int *d_arr;
    hipMalloc(&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    bubble_sort<<<32, 512>>>(d_arr, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Vreme izvršenja: %f ms\n", milliseconds);

    hipFree(d_arr);
    free(h_arr);
    return 0;
}
