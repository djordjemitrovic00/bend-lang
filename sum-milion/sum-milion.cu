#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sum_reduce(int* input, unsigned long long* output, int n) {
    extern __shared__ int shared_data[];

    int tid = threadIdx.x;
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = (global_id < n) ? input[global_id] : 0;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, (unsigned long long)shared_data[0]);
    }
}

int main() {
    int n = 1000000;
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    int* h_input = (int*)malloc(n * sizeof(int));
    unsigned long long h_output = 0;

    for (int i = 0; i < n; i++) {
        h_input[i] = i + 1;
    }

    int* d_input;
    unsigned long long* d_output;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    
    hipMalloc((void**)&d_input, n * sizeof(int));
    hipMalloc((void**)&d_output, sizeof(unsigned long long));

    hipMemcpy(d_input, h_input, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output, &h_output, sizeof(unsigned long long), hipMemcpyHostToDevice);

    sum_reduce<<<grid_size, block_size, block_size * sizeof(int)>>>(d_input, d_output, n);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    hipMemcpy(&h_output, d_output, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    printf("Suma prvih %d prirodnih brojeva je: %llu\n", n, h_output);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Vreme izvršenja na GPU: %f ms\n", milliseconds);

    free(h_input);
    hipFree(d_input);
    hipFree(d_output);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
